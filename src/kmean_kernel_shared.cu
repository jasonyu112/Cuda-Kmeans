#include "hip/hip_runtime.h"
#include "kmeans_kernel.h"

int kmeans_cuda_shared(options_t * opts){
    hipEvent_t wholeStart, wholeStop, memcpyWholeStart, memcpyWholeStop;
    hipEventCreate(&wholeStart);
    hipEventCreate(&wholeStop);
    hipEventCreate(&memcpyWholeStart);
    hipEventCreate(&memcpyWholeStop);
    float total_execution_time = 0;
    float total_memcpy_time = 0;
    float temp_total_memcpy_time;
    hipEventRecord(wholeStart);

    //initialize and read inputs
    std::ifstream in;
    in.open(opts->in_file);
    int size = 0;   
    in >> size;
    in.close();
    int dims = opts->dims;
    int k = opts->num_clusters;
    double* input = (double*)malloc(size*dims*sizeof(double));
    double* center = (double*)malloc(k*dims*sizeof(double));

    read_file(opts, &size, input);
    
    //randomly generating centroids
    kmeans_srand(opts->seed);
    for(int i = 0;i<k;i++){
        int index = kmeans_rand()% size;
        hipEventRecord(memcpyWholeStart);
        std::memcpy(center+i*dims, input+index*dims, dims*sizeof(double));
        hipEventRecord(memcpyWholeStop);
        hipEventSynchronize(memcpyWholeStop);
        hipEventElapsedTime(&temp_total_memcpy_time, memcpyWholeStart, memcpyWholeStop);
        total_memcpy_time+=temp_total_memcpy_time;
    }

    //setup cuda device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp,dev));
    CHECK(hipSetDevice(dev));
    /*
    int blockSize;   // The launch block size
    int minGridSize; // The minimum grid size needed to achieve maximum occupancy
    int gridSize;    // The actual grid size to use, based on the device

    // Find the optimal launch parameters for maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, mapDataShared, 0, 0);

    // Calculate launch grid size and shared memory
    gridSize = (deviceProp.multiProcessorCount * blockSize + deviceProp.warpSize - 1) / deviceProp.warpSize;

    printf("Optimal Block Size: %d\n", blockSize);
    printf("Grid Size: %d\n", gridSize);

    int activeWarpsPerSM;
    int maxWarpsPerSM = deviceProp.maxThreadsPerMultiProcessor / deviceProp.warpSize;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&activeWarpsPerSM, avgData, 32, 0);
    float occupancy = (float)activeWarpsPerSM / maxWarpsPerSM;
    printf("Occupancy: %.2f%%\n", occupancy * 100.0f);
    */
    /*
    printf("Number of multiprocessors: %d\n", deviceProp.multiProcessorCount);
    printf("Clock Rate (kHz): %d\n", deviceProp.clockRate);
    printf("Total amount of constant memory: %4.2f KB\n",deviceProp.totalConstMem/1024.0);
    printf("Total amount of shared memory per block: %4.2f KB\n",deviceProp.sharedMemPerBlock/1024.0);
    printf("Total number of registers available per block: %d\n",deviceProp.regsPerBlock);
    printf("Warp size: %d\n", deviceProp.warpSize);
    printf("Maximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);
    printf("Maximum number of threads per multiprocessor: %d\n",deviceProp.maxThreadsPerMultiProcessor);
    printf("Maximum number of warps per multiprocessor: %d\n",deviceProp.maxThreadsPerMultiProcessor/32);
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(mapDataShared));
    printf("Shared memory per block: %zu bytes\n", attr.sharedSizeBytes);
    */
    //int blockSize = 32;
    //int maxActiveBlocks =0;
    //hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, mapDataShared, blockSize, 0);
    //int blockCount = 68 * maxActiveBlocks;
    //printf("gridsize: %d\n", blockCount);
    
    //setup matrix size
    int input_x = size;
    int input_y = dims;
    int input_xy = input_x*input_y;
    int inputBytes = input_xy *sizeof(double);
    int centroid_x = k;
    int centroid_y = dims;
    int centroid_xy = centroid_x*centroid_y;
    int centroidBytes = centroid_xy*sizeof(double);
    int no_c_x = size;
    int no_c_Bytes = no_c_x*sizeof(int);
    int labels_Bytes = sizeof(double)*(k*(dims+1));
    double* device_inputs;
    double* device_centroids;
    int* device_no_c;
    double* device_labels;
    double* device_oldCentroid;
    int* device_converged;

    hipMalloc((void**)&device_inputs, inputBytes);
    hipMalloc((void**)&device_centroids, centroidBytes);
    hipMalloc((void**)&device_no_c, no_c_Bytes);
    hipMalloc((void**)&device_labels, labels_Bytes);
    hipMalloc((void**)&device_oldCentroid, centroidBytes);
    hipMalloc((void**)&device_converged, sizeof(int));

    //transfer inputs to device
    hipEventRecord(memcpyWholeStart);
    hipMemcpy(device_inputs, input, inputBytes, hipMemcpyHostToDevice);
    hipEventRecord(memcpyWholeStop);
    hipEventSynchronize(memcpyWholeStop);
    hipEventElapsedTime(&temp_total_memcpy_time, memcpyWholeStart, memcpyWholeStop);
    total_memcpy_time+=temp_total_memcpy_time;
    
    int iterations = 0;
    int done = opts->max_iter;
    int no_c[size];
    float memcpyTime = 0;
    std::vector<float*> timeVecs;

    hipEventRecord(memcpyWholeStart);
    hipMemcpy(device_centroids, center, centroidBytes, hipMemcpyHostToDevice);
    hipMemcpy(device_oldCentroid, center, centroidBytes, hipMemcpyHostToDevice);
    hipEventRecord(memcpyWholeStop);
    hipEventSynchronize(memcpyWholeStop);
    hipEventElapsedTime(&temp_total_memcpy_time, memcpyWholeStart, memcpyWholeStop);
    total_memcpy_time+=temp_total_memcpy_time;

    hipEvent_t start, stop, loop_memStart, loop_memStop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&loop_memStart);
    hipEventCreate(&loop_memStop);
    while(true){
        hipEventRecord(start);
        double labels[k*(dims+1)] = {};
        hipEventRecord(loop_memStart);
        hipMemcpy(device_labels, labels, labels_Bytes, hipMemcpyHostToDevice);
        hipEventRecord(loop_memStop);
        hipEventSynchronize(loop_memStop);
        float loop_memTime;
        hipEventElapsedTime(&loop_memTime, loop_memStart, loop_memStop);
        memcpyTime+=loop_memTime;

        int block = 32;
        //int grid = 1088;
        int grid = (size+block-1)/block;
        int converged = 1;

        hipEventRecord(loop_memStart);
        hipMemcpy(device_converged, &converged, sizeof(int), hipMemcpyHostToDevice);
        hipEventRecord(loop_memStop);
        hipEventSynchronize(loop_memStop);
        hipEventElapsedTime(&loop_memTime, loop_memStart, loop_memStop);
        memcpyTime+=loop_memTime;

        int sharedMemSize = (k * (dims+1)) * sizeof(double);
        mapDataShared<<< grid, block,sharedMemSize >>>(device_inputs, device_centroids, device_labels, device_no_c, size, dims, k);

        block = 32;
        grid = ((k*dims)+block-1)/block;
        avgData<<< grid, block>>>(device_centroids, device_labels, dims, k);
        checkConvergence<<< grid, block>>>(device_oldCentroid, device_centroids, k, dims, opts->threshhold,device_converged);

        hipEventRecord(loop_memStart);
        hipMemcpy(&converged, device_converged, sizeof(int), hipMemcpyDeviceToHost);
        hipEventRecord(loop_memStop);
        hipEventSynchronize(loop_memStop);
        hipEventElapsedTime(&loop_memTime, loop_memStart, loop_memStop);
        memcpyTime+=loop_memTime;

        iterations+=1;
        if(converged || iterations>=done){
            hipEventRecord(loop_memStart);
            hipMemcpy(center, device_centroids, centroidBytes, hipMemcpyDeviceToHost);
            if(opts->control == false){
                hipMemcpy(no_c, device_no_c, no_c_Bytes, hipMemcpyDeviceToHost);
            }
            hipEventRecord(loop_memStop);
            hipEventSynchronize(loop_memStop);
            hipEventElapsedTime(&loop_memTime, loop_memStart, loop_memStop);
            memcpyTime+=loop_memTime;

            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float* time = (float*)malloc(sizeof(float));
            hipEventElapsedTime(time, start, stop);
            timeVecs.push_back(time);
            double time_per_iter_in_ms = 0;
            for(unsigned int i =0;i<timeVecs.size();i++){
                time_per_iter_in_ms+=*timeVecs[i];
            }
            time_per_iter_in_ms= time_per_iter_in_ms/timeVecs.size();
            printf("%d,%lf\n", iterations, time_per_iter_in_ms);
            if(opts->control == false){
                printf("clusters:");
                for (int p=0; p < size; p++)
                    printf(" %d", no_c[p]);
            }else{
                for (int clusterId = 0; clusterId < k; clusterId ++){
                    printf("%d ", clusterId);
                    for (int d = 0; d < dims; d++)
                        printf("%lf ", center[clusterId*dims+d]);
                    printf("\n");
                }
            }
            break;
        }
        else{
            hipEventRecord(loop_memStart);
            hipMemcpy(device_oldCentroid, device_centroids, centroidBytes, hipMemcpyDeviceToDevice);
            hipEventRecord(loop_memStop);
            hipEventSynchronize(loop_memStop);
            hipEventElapsedTime(&loop_memTime, loop_memStart, loop_memStop);
            memcpyTime+=loop_memTime;
        }
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float* time = (float*)malloc(sizeof(float));
        hipEventElapsedTime(time, start, stop);
        timeVecs.push_back(time);
    }
    hipEventRecord(wholeStop);
    hipEventSynchronize(wholeStop);
    hipEventElapsedTime(&total_execution_time, wholeStart, wholeStop);

    hipFree(device_inputs);
    hipFree(device_centroids);
    hipFree(device_no_c);
    hipFree(device_labels);
    hipFree(device_oldCentroid);
    hipFree(device_converged);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(loop_memStart);
    hipEventDestroy(loop_memStop);
    hipEventDestroy(wholeStart);
    hipEventDestroy(wholeStop);
    hipEventDestroy(memcpyWholeStart);
    hipEventDestroy(memcpyWholeStop);
    for(int i = 0; i<timeVecs.size();i++){
        free(timeVecs[i]);
    }
    free(input);
    free(center);
    total_memcpy_time += memcpyTime;
    printf("%lf,%lf,%lf\n", memcpyTime,total_memcpy_time,total_execution_time);
    return 0;
}

__global__ void mapDataShared(double* input, double* centroids, double* labels, int* no_c, int size, int dims, int k) {
    extern __shared__ double s_centroids[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // each thread loads a part of the centroids into shared memory
    #pragma unroll
    for (int i = tid; i < k * dims; i ++) {
        s_centroids[i] = centroids[i];
    }

    __syncthreads();

    if (idx < size) {
        double smallest_distance = DBL_MAX;
        int new_index = -1;

        for (int k_index = 0; k_index < k; k_index++) {
            double e_dist = 0;
            for (int j = 0; j < dims; j++) {
                double abs_diff = input[idx*dims+j]-s_centroids[k_index*dims+j];
                e_dist += abs_diff*abs_diff;
            }
            e_dist = e_dist;
            if (e_dist < smallest_distance) {
                smallest_distance = e_dist;
                new_index = k_index;
            }
        }

        for (int j = 0; j < dims; j++) {
            atomicAdd(&labels[new_index*(dims+1)+j], input[idx*dims+j]);
        }
        atomicAdd(&labels[new_index*(dims+1)+dims], 1);
        no_c[idx] = new_index;
    }
}

/*
__global__ void avgDataShared(double* centroids, double* labels, int dims, int k){
    extern __shared__ double sharedMem[];

    double* s_labels = sharedMem;

    int centroid_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    for(int i = tid; i<k*(dims+1);i++){
        s_labels[i]= labels[i];
    }
    __syncthreads();

    if (centroid_idx < k * dims){
        int k_index = centroid_idx / dims;
        int dim_index = centroid_idx %dims;
        int count = s_labels[k_index*(dims + 1) + dims];
        atomicExch((unsigned long long int*)&centroids[k_index * (dims)+dim_index], __double_as_longlong(s_labels[k_index*(dims+1)+dim_index]/count));
    }
}

__global__ void checkConvergenceShared(double* oldCentroids, double* centroids, int k, int dims, double threshhold, int* converged){
    //check for convergence between centroids and oldCentroids
    extern __shared__ double sharedMem[];

    double* s_oldCentroids = sharedMem;
    double* s_centroids = sharedMem+dims*k;

    int tid = threadIdx.x;

    for(int i = tid;i<k*dims;i++){
        s_oldCentroids[i] = oldCentroids[i];
        s_centroids[i] = centroids[i];
    }
    __syncthreads();

    int centroid_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(centroid_idx<k* dims){
        int k_index = centroid_idx / dims;
        int dim_index = centroid_idx%dims;
        double abs_diff = fabs(s_centroids[k_index*dims+dim_index]-s_oldCentroids[k_index*dims+dim_index]);
        if(abs_diff>(threshhold*threshhold)){
            *converged = 0;
        }
    }
}
*/